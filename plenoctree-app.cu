#include "hip/hip_runtime.h"
#include <torch/script.h> // One-stop header.
#include <torch/torch.h>
#include <iostream>

#include "data_spec.hpp"
#include "common.cuh"
#include "data_spec_packed.cuh"

namespace {

// Automatically choose number of CUDA threads based on HW CUDA kernel count
int cuda_n_threads = -1;
__host__ void auto_cuda_threads() {
    if (~cuda_n_threads) return;
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0);
    const int n_cores = get_sp_cores(dev_prop);
    // Optimize number of CUDA threads per block
    if (n_cores < 2048) {
        cuda_n_threads = 256;
    } if (n_cores < 8192) {
        cuda_n_threads = 512;
    } else {
        cuda_n_threads = 1024;
    }
}

namespace device {
// SH Coefficients from https://github.com/google/spherical-harmonics
__device__ __constant__ const float C0 = 0.28209479177387814;
__device__ __constant__ const float C1 = 0.4886025119029199;
__device__ __constant__ const float C2[] = {
    1.0925484305920792,
    -1.0925484305920792,
    0.31539156525252005,
    -1.0925484305920792,
    0.5462742152960396
};

__device__ __constant__ const float C3[] = {
    -0.5900435899266435,
    2.890611442640554,
    -0.4570457994644658,
    0.3731763325901154,
    -0.4570457994644658,
    1.445305721320277,
    -0.5900435899266435
};

__device__ __constant__ const float C4[] = {
    2.5033429417967046,
    -1.7701307697799304,
    0.9461746957575601,
    -0.6690465435572892,
    0.10578554691520431,
    -0.6690465435572892,
    0.47308734787878004,
    -1.7701307697799304,
    0.6258357354491761,
};

#define _SOFTPLUS_M1(x) (logf(1 + expf((x) - 1)))
#define _SIGMOID(x) (1 / (1 + expf(-(x))))

template<typename scalar_t>
__host__ __device__ __inline__ static scalar_t _norm(
                scalar_t* dir) {
    return sqrtf(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
}

template<typename scalar_t>
__host__ __device__ __inline__ static void _normalize(
                scalar_t* dir) {
    scalar_t norm = _norm(dir);
    dir[0] /= norm; dir[1] /= norm; dir[2] /= norm;
}

template<typename scalar_t>
__host__ __device__ __inline__ static scalar_t _dot3(
        const scalar_t* __restrict__ u,
        const scalar_t* __restrict__ v) {
    return u[0] * v[0] + u[1] * v[1] + u[2] * v[2];
}

// Calculate basis functions depending on format, for given view directions
template <typename scalar_t>
__device__ __inline__ void maybe_precalc_basis(
    const int format,
    const int basis_dim,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>
        extra,
    const scalar_t* __restrict__ dir,
    scalar_t* __restrict__ out) {
    switch(format) {
        case FORMAT_ASG:
            {
                // UNTESTED ASG
                for (int i = 0; i < basis_dim; ++i) {
                    const auto& ptr = extra[i];
                    scalar_t S = _dot3(dir, &ptr[8]);
                    scalar_t dot_x = _dot3(dir, &ptr[2]);
                    scalar_t dot_y = _dot3(dir, &ptr[5]);
                    out[i] = S * expf(-ptr[0] * dot_x * dot_x
                                      -ptr[1] * dot_y * dot_y) / basis_dim;
                }
            }  // ASG
            break;
        case FORMAT_SG:
            {
                for (int i = 0; i < basis_dim; ++i) {
                    const auto& ptr = extra[i];
                    out[i] = expf(ptr[0] * (_dot3(dir, &ptr[1]) - 1.f)) / basis_dim;
                }
            }  // SG
            break;
        case FORMAT_SH:
            {
                out[0] = C0;
                const scalar_t x = dir[0], y = dir[1], z = dir[2];
                const scalar_t xx = x * x, yy = y * y, zz = z * z;
                const scalar_t xy = x * y, yz = y * z, xz = x * z;
                switch (basis_dim) {
                    case 25:
                        out[16] = C4[0] * xy * (xx - yy);
                        out[17] = C4[1] * yz * (3 * xx - yy);
                        out[18] = C4[2] * xy * (7 * zz - 1.f);
                        out[19] = C4[3] * yz * (7 * zz - 3.f);
                        out[20] = C4[4] * (zz * (35 * zz - 30) + 3);
                        out[21] = C4[5] * xz * (7 * zz - 3);
                        out[22] = C4[6] * (xx - yy) * (7 * zz - 1.f);
                        out[23] = C4[7] * xz * (xx - 3 * yy);
                        out[24] = C4[8] * (xx * (xx - 3 * yy) - yy * (3 * xx - yy));
                        [[fallthrough]];
                    case 16:
                        out[9] = C3[0] * y * (3 * xx - yy);
                        out[10] = C3[1] * xy * z;
                        out[11] = C3[2] * y * (4 * zz - xx - yy);
                        out[12] = C3[3] * z * (2 * zz - 3 * xx - 3 * yy);
                        out[13] = C3[4] * x * (4 * zz - xx - yy);
                        out[14] = C3[5] * z * (xx - yy);
                        out[15] = C3[6] * x * (xx - 3 * yy);
                        [[fallthrough]];
                    case 9:
                        out[4] = C2[0] * xy;
                        out[5] = C2[1] * yz;
                        out[6] = C2[2] * (2.0 * zz - xx - yy);
                        out[7] = C2[3] * xz;
                        out[8] = C2[4] * (xx - yy);
                        [[fallthrough]];
                    case 4:
                        out[1] = -C1 * y;
                        out[2] = C1 * z;
                        out[3] = -C1 * x;
                }
            }  // SH
            break;

        default:
            // Do nothing
            break;
    }  // switch
}

template <typename scalar_t>
__device__ __inline__ scalar_t _get_delta_scale(
    const scalar_t* __restrict__ scaling,
    scalar_t* __restrict__ dir) {
    dir[0] *= scaling[0];
    dir[1] *= scaling[1];
    dir[2] *= scaling[2];
    scalar_t delta_scale = 1.f / _norm(dir);
    dir[0] *= delta_scale;
    dir[1] *= delta_scale;
    dir[2] *= delta_scale;
    return delta_scale;
}

template <typename scalar_t>
__device__ __inline__ void _dda_unit(
        const scalar_t* __restrict__ cen,
        const scalar_t* __restrict__ invdir,
        scalar_t* __restrict__ tmin,
        scalar_t* __restrict__ tmax) {
    // Intersect unit AABB
    scalar_t t1, t2;
    *tmin = 0.0f;
    *tmax = 1e9f;
#pragma unroll
    for (int i = 0; i < 3; ++i) {
        t1 = - cen[i] * invdir[i];
        t2 = t1 +  invdir[i];
        *tmin = max(*tmin, min(t1, t2));
        *tmax = min(*tmax, max(t1, t2));
    }
}

template <typename scalar_t>
__device__ __inline__ void trace_ray(
        PackedTreeSpec<scalar_t>& __restrict__ tree,
        SingleRaySpec<scalar_t> ray,
        RenderOptions& __restrict__ opt,
        torch::TensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, int32_t> out) {
    const scalar_t delta_scale = _get_delta_scale(tree.scaling, ray.dir);

    scalar_t tmin, tmax;
    scalar_t invdir[3];
    const int tree_N = tree.child.size(1);
    const int data_dim = tree.data.size(4);
    const int out_data_dim = out.size(0);

// https://blog.csdn.net/AMDS123/article/details/79541481
#pragma unroll
    for (int i = 0; i < 3; ++i) {
        invdir[i] = 1.0 / (ray.dir[i] + 1e-9);
    }
    _dda_unit(ray.origin, invdir, &tmin, &tmax);

    if (tmax < 0 || tmin > tmax) {
        // Ray doesn't hit box
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = opt.background_brightness;
        }
        return;
    } else {
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] = 0.f;
        }
        scalar_t pos[3];
        scalar_t basis_fn[25];
        maybe_precalc_basis<scalar_t>(opt.format, opt.basis_dim,
                tree.extra_data, ray.vdir, basis_fn);

        scalar_t light_intensity = 1.f;
        scalar_t t = tmin;
        scalar_t cube_sz;
        const scalar_t d_rgb_pad = 1 + 2 * opt.rgb_padding;
        while (t < tmax) {
            for (int j = 0; j < 3; ++j) {
                pos[j] = ray.origin[j] + t * ray.dir[j];
            }

            int64_t node_id;
            scalar_t* tree_val = query_single_from_root<scalar_t>(tree.data, tree.child,
                        pos, &cube_sz, tree.weight_accum != nullptr ? &node_id : nullptr);

            scalar_t att;
            scalar_t subcube_tmin, subcube_tmax;
            _dda_unit(pos, invdir, &subcube_tmin, &subcube_tmax);

            const scalar_t t_subcube = (subcube_tmax - subcube_tmin) / cube_sz;
            const scalar_t delta_t = t_subcube + opt.step_size;
            scalar_t sigma = tree_val[data_dim - 1];
            if (opt.density_softplus) sigma = _SOFTPLUS_M1(sigma);
            if (sigma > opt.sigma_thresh) {
                att = expf(-delta_t * delta_scale * sigma);
                const scalar_t weight = light_intensity * (1.f - att);

                if (opt.format != FORMAT_RGBA) {
                    for (int t = 0; t < out_data_dim; ++ t) {
                        int off = t * opt.basis_dim;
                        scalar_t tmp = 0.0;
                        for (int i = opt.min_comp; i <= opt.max_comp; ++i) {
                            tmp += basis_fn[i] * tree_val[off + i];
                        }
                        out[t] += weight * (_SIGMOID(tmp) * d_rgb_pad - opt.rgb_padding);
                    }
                } else {
                    for (int j = 0; j < out_data_dim; ++j) {
                        out[j] += weight * (_SIGMOID(tree_val[j]) * d_rgb_pad - opt.rgb_padding);
                    }
                }
                light_intensity *= att;

                if (tree.weight_accum != nullptr) {
                    if (tree.weight_accum_max) {
                        atomicMax(&tree.weight_accum[node_id], weight);
                    } else {
                        atomicAdd(&tree.weight_accum[node_id], weight);
                    }
                }

                if (light_intensity <= opt.stop_thresh) {
                    // Full opacity, stop
                    scalar_t scale = 1.0 / (1.0 - light_intensity);
                    for (int j = 0; j < out_data_dim; ++j) {
                        out[j] *= scale;
                    }
                    return;
                }
            }
            t += delta_t;
        }
        for (int j = 0; j < out_data_dim; ++j) {
            out[j] += light_intensity * opt.background_brightness;
        }
    }
}

template <typename scalar_t>
__device__ __inline__ void cam2world_ray(
    int ix, int iy,
    scalar_t* dir,
    scalar_t* origin,
    const PackedCameraSpec<scalar_t>& __restrict__ cam) {
    scalar_t x = (ix - 0.5 * cam.width) / cam.fx;
    scalar_t y = -(iy - 0.5 * cam.height) / cam.fy;
    scalar_t z = sqrtf(x * x + y * y + 1.0);
    x /= z; y /= z; z = -1.0f / z;
    dir[0] = cam.c2w[0][0] * x + cam.c2w[0][1] * y + cam.c2w[0][2] * z;
    dir[1] = cam.c2w[1][0] * x + cam.c2w[1][1] * y + cam.c2w[1][2] * z;
    dir[2] = cam.c2w[2][0] * x + cam.c2w[2][1] * y + cam.c2w[2][2] * z;
    origin[0] = cam.c2w[0][3]; origin[1] = cam.c2w[1][3]; origin[2] = cam.c2w[2][3];
}


template <typename scalar_t>
__host__ __device__ __inline__ static void maybe_world2ndc(
        RenderOptions& __restrict__ opt,
        scalar_t* __restrict__ dir,
        scalar_t* __restrict__ cen, scalar_t near = 1.f) {
    if (opt.ndc_width < 0)
        return;
    scalar_t t = -(near + cen[2]) / dir[2];
    for (int i = 0; i < 3; ++i) {
        cen[i] = cen[i] + t * dir[i];
    }

    dir[0] = -((2 * opt.ndc_focal) / opt.ndc_width) * (dir[0] / dir[2] - cen[0] / cen[2]);
    dir[1] = -((2 * opt.ndc_focal) / opt.ndc_height) * (dir[1] / dir[2] - cen[1] / cen[2]);
    dir[2] = -2 * near / cen[2];

    cen[0] = -((2 * opt.ndc_focal) / opt.ndc_width) * (cen[0] / cen[2]);
    cen[1] = -((2 * opt.ndc_focal) / opt.ndc_height) * (cen[1] / cen[2]);
    cen[2] = 1 + 2 * near / cen[2];

    _normalize(dir);
}

template <typename scalar_t>
__global__ void render_image_kernel(
    PackedTreeSpec<scalar_t> tree,
    PackedCameraSpec<scalar_t> cam,
    RenderOptions opt,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits>
        out) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    scalar_t dir[3], origin[3];
    cam2world_ray(ix, iy, dir, origin, cam);
    scalar_t vdir[3] = {dir[0], dir[1], dir[2]};
    maybe_world2ndc(opt, dir, origin);

    transform_coord<scalar_t>(origin, tree.offset, tree.scaling);
    trace_ray<scalar_t>(
        tree,
        SingleRaySpec<scalar_t>{origin, dir, vdir},
        opt,
        out[iy][ix]);
}

}  // namespace device


// Compute RGB output dimension from input dimension & SH degree
__host__ int get_out_data_dim(int format, int basis_dim, int in_data_dim) {
    if (format != FORMAT_RGBA) {
        return (in_data_dim - 1) / basis_dim;
    } else {
        return in_data_dim - 1;
    }
}

}  // namespace

#define PI 3.141592653589793

using torch::Tensor;


torch::Tensor trans_t(double t) {
  return torch::tensor({{1., 0., 0., 0.}, \
                        {0., 1., 0., 0.}, \
                        {0., 0., 1., t }, \
                        {0., 0., 0., 1.}});
}

torch::Tensor rot_phi(double phi) {
  return torch::tensor({{1., 0., 0., 0.}, \
                        {0., std::cos(phi),-std::sin(phi), 0.}, \
                        {0., std::sin(phi), std::cos(phi), 0.}, \
                        {0., 0., 0., 1.}});
}

torch::Tensor rot_theta(double th) {
  return torch::tensor({{std::cos(th),0.,-std::sin(th), 0.}, \
                        {0., 1., 0., 0.}, \
                        {std::sin(th),0., std::cos(th), 0.}, \
                        {0., 0., 0., 1.}});
}

torch::Tensor pose_spherical(double theta, double phi, double radius) {
  torch::Tensor c2w = trans_t(radius);
  c2w = torch::matmul(rot_phi(phi/180.*PI), c2w);
  c2w = torch::matmul(rot_theta(theta/180.*PI), c2w);
  c2w = torch::matmul(torch::tensor({{-1., 0., 0., 0.}, {0., 0., 1., 0.}, \
                      {0., 1., 0., 0.}, {0., 0., 0., 1.}}), c2w);
  return c2w;
}


torch::Tensor volume_render_image(TreeSpec& tree, CameraSpec& cam, RenderOptions& opt) {
    tree.check();
    cam.check();
    DEVICE_GUARD(tree.data); // Set the current CUDA device to the passed Device
    const size_t Q = size_t(cam.width) * cam.height;

    auto_cuda_threads();
    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, cuda_n_threads);
    int out_data_dim = get_out_data_dim(opt.format, opt.basis_dim, tree.data.size(4));
    torch::Tensor result = torch::zeros({cam.height, cam.width, out_data_dim},
            tree.data.options()); // TensorOptions

    // https://zhuanlan.zhihu.com/p/48463543
    // https://pytorch.org/cppdocs/notes/tensor_basics.html
    AT_DISPATCH_FLOATING_TYPES(tree.data.type(), __FUNCTION__, [&] {
            device::render_image_kernel<scalar_t><<<blocks, cuda_n_threads>>>(
                    tree, cam, opt,
                    result.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>());
    });
    CUDA_CHECK_ERRORS;
    return result;
}


int main() {
  

  torch::jit::script::Module tree_spec_dict = torch::jit::load("tree_spec_dict.pt", torch::kCUDA);
  //tree_spec_dict.to(at::kCUDA);

  TreeSpec tree_spec;
  tree_spec.data = tree_spec_dict.attr("data").toTensor();
  tree_spec.child = tree_spec_dict.attr("child").toTensor();
  tree_spec.parent_depth = tree_spec_dict.attr("parent_depth").toTensor();
  tree_spec.extra_data = tree_spec_dict.attr("extra_data").toTensor();
  tree_spec.offset = tree_spec_dict.attr("offset").toTensor();
  tree_spec.scaling = tree_spec_dict.attr("scaling").toTensor();
  tree_spec._weight_accum = tree_spec_dict.attr("_weight_accum").toTensor();
  tree_spec._weight_accum_max = tree_spec_dict.attr("_weight_accum_max").toBool();

  torch::jit::script::Module options_dict = torch::jit::load("options_dict.pt", torch::kCUDA);

  RenderOptions options;
  options.step_size = options_dict.attr("step_size").toDouble();
  options.background_brightness = options_dict.attr("background_brightness").toDouble();
  options.format = options_dict.attr("format").toInt();
  options.basis_dim = options_dict.attr("basis_dim").toInt();
  options.ndc_width = options_dict.attr("ndc_width").toInt();
  options.ndc_height = options_dict.attr("ndc_height").toInt();
  options.ndc_focal = options_dict.attr("ndc_focal").toDouble();
  options.min_comp = options_dict.attr("min_comp").toInt();
  options.max_comp = options_dict.attr("max_comp").toInt();
  options.sigma_thresh = options_dict.attr("sigma_thresh").toDouble();
  options.stop_thresh = options_dict.attr("stop_thresh").toDouble();
  options.density_softplus = options_dict.attr("density_softplus").toBool();
  options.rgb_padding = options_dict.attr("rgb_padding").toDouble();

  auto c2w = pose_spherical(90,-30,4);
  std::cout << c2w << std::endl;
  
  CameraSpec camera_spec;
  camera_spec.c2w = c2w.to(torch::device(torch::kCUDA));
  camera_spec.fx = 1111.111;
  camera_spec.fy = 1111.111;
  camera_spec.width = 800;
  camera_spec.height = 800;

  auto result = volume_render_image(tree_spec, camera_spec, options);
  
  std::cout << result.sizes() << std::endl;

  torch::save({result.to(torch::device(torch::kCPU))}, "rgb_map.pt");
}
